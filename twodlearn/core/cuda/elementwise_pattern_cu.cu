#include "hip/hip_runtime.h"
//  ***********************************************************************
//  Description:   GPU implementation of element-wise pattern operations
//  Wrote by: Daniel L. Marino (marinodl@vcu.edu) (2016)
//  ***********************************************************************
#include "elementwise_pattern_cu.h"

// look at http://docs.nvidia.com/cuda/thrust/#axzz4KT0tW3IU to see examples of how to use thurst


template <typename FunctorOp, typename T, int BLOCK_SIZE>
__global__ void elementwise_pattern_cuda(T* in_tensor, int length, FunctorOp elementwise_op){
  // Block index
  int bx = blockIdx.x;

  // Thread index
  int tx = threadIdx.x;

  // create functor
  //FunctorOp elementwise_op;

  // get element index
  int a_idx = bx*BLOCK_SIZE + tx;

  // apply functor
  if (a_idx<length)
    in_tensor[a_idx] = elementwise_op( in_tensor[a_idx] );
}



template <typename FunctorOp, typename T, int BLOCK_SIZE>
__global__ void elementwise_pattern_cuda(T* out_tensor, T* in_tensor, int length, FunctorOp elementwise_op){

  // Block index
  int bx = blockIdx.x;

  // Thread index
  int tx = threadIdx.x;

  // create functor
  //FunctorOp elementwise_op;

  // get element index
  int a_idx = bx*BLOCK_SIZE + tx;

  // apply functor
  if (a_idx<length)
    out_tensor[a_idx] = elementwise_op( in_tensor[a_idx] );

}


template <typename FunctorOp, typename T, int BLOCK_SIZE>
__global__ void elementwise_pattern_cuda(T* out_tensor, T* in_tensor1, T* in_tensor2, int length, FunctorOp elementwise_op){
  // Block index
  int bx = blockIdx.x;

  // Thread index
  int tx = threadIdx.x;

  // create functor
  //FunctorOp elementwise_op;

  // get element index
  int a_idx = bx*BLOCK_SIZE + tx;

  // apply functor
  if (a_idx<length)
    out_tensor[a_idx] = elementwise_op( in_tensor1[a_idx], in_tensor2[a_idx] );
}
